#include "hip/hip_runtime.h"
#include <iostream>
#include <sstream>
#include <vector>

#include "main_runner.h"

hipError_t CutlassSgemmNN(
  int M,
  int N,
  int K,
  float *A,
  int lda,
  float *B,
  int ldb,
  float *C,
  int ldc) {
  Array<int, 3> arr;
  arr[0] = N; arr[1] = M; arr[2] = K;
  auto args = Params(
      arr,
      B, A, C, C,
      ldb, lda, ldc, ldc);
  MainRunner gemm_operator;
  bool status = gemm_operator(args);
  if (!status) {
    return hipErrorUnknown;
  }
  return hipSuccess;
}

__global__ void InitializeMatrix_kernel(
  float *matrix,
  int ldm,
  int rows,
  int columns,
  int seed = 0) {

  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;

  if (i < rows && j < columns) {
    int offset = i + j * ldm;

    // Generate arbitrary elements.
    int const k = 16807;
    int const m = 16;
    float value = float(((offset + seed) * k % m) - m / 2);

    matrix[offset] = value;
  }
}

hipError_t InitializeMatrix(float *matrix, int ldm, int rows, int columns, int seed = 0) {

  dim3 block(16, 16);
  dim3 grid(
    (rows + block.x - 1) / block.x,
    (columns + block.y - 1) / block.y
  );

  InitializeMatrix_kernel<<< grid, block >>>(matrix, ldm, rows, columns, seed);

  return hipGetLastError();
}

hipError_t AllocateMatrix(float **matrix, int ldm, int rows, int columns, int seed = 0) {
  hipError_t result;

  size_t sizeof_matrix = sizeof(float) * ldm * columns;

  // Allocate device memory.
  result = hipMalloc(reinterpret_cast<void **>(matrix), sizeof_matrix);

  if (result != hipSuccess) {
    std::cerr << "Failed to allocate matrix: "
      << hipGetErrorString(result) << std::endl;
    return result;
  }

  // Clear the allocation.
  result = hipMemset(*matrix, 0, sizeof_matrix);

  if (result != hipSuccess) {
    std::cerr << "Failed to clear matrix device memory: "
      << hipGetErrorString(result) << std::endl;
    return result;
  }

  // Initialize matrix elements to arbitrary small integers.
  result = InitializeMatrix(*matrix, ldm, rows, columns, seed);

  if (result != hipSuccess) {
    std::cerr << "Failed to initialize matrix: "
      << hipGetErrorString(result) << std::endl;
    return result;
  }

  return result;
}

__global__ void ReferenceGemm_kernel(
  int M,
  int N,
  int K,
  float alpha,
  float const *A,
  int lda,
  float const *B,
  int ldb,
  float beta,
  float *C,
  int ldc) {

  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;

  if (i < M && j < N) {
    float accumulator = 0;

    for (int k = 0; k < K; ++k) {
      accumulator += A[i + k * lda] * B[k + j * ldb];
    }

    C[i + j * ldc] = alpha * accumulator + beta * C[i + j * ldc];
  }
}

hipError_t ReferenceGemm(
  int M,
  int N,
  int K,
  float alpha,
  float const *A,
  int lda,
  float const *B,
  int ldb,
  float beta,
  float *C,
  int ldc) {

  dim3 block(16, 16);
  dim3 grid(
    (M + block.x - 1) / block.x,
    (N + block.y - 1) / block.y
  );

  ReferenceGemm_kernel<<< grid, block >>>(M, N, K, alpha, A, lda, B, ldb, beta, C, ldc);

  return hipGetLastError();
}

hipError_t TestCutlassGemm(int M, int N, int K, float alpha, float beta) {
  hipError_t result;

  //
  // Define several matrices to be used as operands to GEMM kernels.
  //

  // Compute leading dimensions for each matrix.
  int lda = M;
  int ldb = K;
  int ldc = M;

  // Compute size in bytes of the C matrix.
  size_t sizeof_C = sizeof(float) * ldc * N;

  // Define pointers to matrices in GPU device memory.
  float *A;
  float *B;
  float *C;
  float *C_reference;

  //
  // Allocate matrices in GPU device memory with arbitrary seeds.
  //

  result = AllocateMatrix(&A, lda, M, K, 0);

  if (result !=  hipSuccess) {
    return result;
  }

  result = AllocateMatrix(&B, ldb, K, N, 17);

  if (result !=  hipSuccess) {
    hipFree(A);
    return result;
  }

  result = AllocateMatrix(&C, ldc, M, N, 101);

  if (result != hipSuccess) {
    hipFree(A);
    hipFree(B);
    return result;
  }

  result = AllocateMatrix(&C_reference, ldc, M, N, 101);

  if (result != hipSuccess) {
    hipFree(A);
    hipFree(B);
    hipFree(C);
    return result;
  }

  result = hipMemcpy(C_reference, C, sizeof_C, hipMemcpyDeviceToDevice);

  if (result != hipSuccess) {
    std::cerr << "Failed to copy C matrix to C_reference: "
      << hipGetErrorString(result) << std::endl;

    hipFree(C_reference);
    hipFree(C);
    hipFree(B);
    hipFree(A);

    return result;
  }

  //
  // Launch CUTLASS GEMM.
  //

  result = CutlassSgemmNN(M, N, K, A, lda, B, ldb, C, ldc);

  if (result != hipSuccess) {
    std::cerr << "CUTLASS GEMM kernel failed: "
      << hipGetErrorString(result) << std::endl;

    hipFree(C_reference);
    hipFree(C);
    hipFree(B);
    hipFree(A);

    return result;
  }

  //
  // Verify.
  //

  // Launch reference GEMM
  result = ReferenceGemm(M, N, K, alpha, A, lda, B, ldb, beta, C_reference, ldc);

  if (result != hipSuccess) {
    std::cerr << "Reference GEMM kernel failed: "
      << hipGetErrorString(result) << std::endl;

    hipFree(C_reference);
    hipFree(C);
    hipFree(B);
    hipFree(A);

    return result;
  }

  // Copy to host and verify equivalence.
  std::vector<float> host(ldc * N, 0);
  std::vector<float> host_reference(ldc * N, 0);

  result = hipMemcpy(host.data(), C, sizeof_C, hipMemcpyDeviceToHost);

  if (result != hipSuccess) {
    std::cerr << "Failed to copy CUTLASS GEMM results: "
      << hipGetErrorString(result) << std::endl;

    hipFree(C_reference);
    hipFree(C);
    hipFree(B);
    hipFree(A);

    return result;
  }

  result = hipMemcpy(host_reference.data(), C_reference, sizeof_C, hipMemcpyDeviceToHost);

  if (result != hipSuccess) {
    std::cerr << "Failed to copy Reference GEMM results: "
      << hipGetErrorString(result) << std::endl;

    hipFree(C_reference);
    hipFree(C);
    hipFree(B);
    hipFree(A);

    return result;
  }

  //
  // Free device memory allocations.
  //

  hipFree(C_reference);
  hipFree(C);
  hipFree(B);
  hipFree(A);

  //
  // Test for bit equivalence of results.
  //

  if (host != host_reference) {
    std::cerr << "CUTLASS results incorrect." << std::endl;

    return hipErrorUnknown;
  } else {
    std::cout << "Passed." << std::endl;
    return hipSuccess;
  }
}

int main() {
  int problem[3] = { 4096, 4096, 4096 };
  float scalars[2] = { 1, 0 };
  hipError_t result = TestCutlassGemm(
    problem[0],     // GEMM M dimension
    problem[1],     // GEMM N dimension
    problem[2],     // GEMM K dimension
    scalars[0],     // alpha
    scalars[1]      // beta
  );

  return result == hipSuccess ? 0 : -1;
}
